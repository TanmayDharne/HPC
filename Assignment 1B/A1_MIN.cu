
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>

using namespace std;

__global__ void mini1(int *a,int *b,int n)
{
	int block=256*blockIdx.x;
	int mini=7888888;

	for(int i=block;i<min(256+block,n);i++)
	{
		if(mini>a[i])
		{
			mini=a[i];
		}
	}
	b[blockIdx.x]=mini;
}

int main()
{
	//cout<<"Enter the size of array"<<endl;
	int n=10;
	//cin>>n;
	int a[10];

	for(int i=0;i<10;i++)
	{
		a[i]=i+1;
	
	}
	
	int *ad,*bd;
	int size=n*sizeof(int);
	
	hipMalloc(&ad,size);
	hipMemcpy(ad,a,size,hipMemcpyHostToDevice);

	int grids=ceil(n*1.0f/256.0f);
	hipMalloc(&bd,grids*sizeof(int));

	dim3 grid(grids,1);
	dim3 block(1,1);

	while(n>1)
	{
		mini1<<<grids,block>>>(ad,bd,n);
		n=ceil(n*1.0f/256.0f);
		hipMemcpy(ad,bd,n*sizeof(int),hipMemcpyDeviceToDevice);
	}

	int ans[2];

	hipMemcpy(ans,ad,4,hipMemcpyDeviceToHost);
	
	cout<<"The minimum element is"<<ans[0]<<endl;
} 
